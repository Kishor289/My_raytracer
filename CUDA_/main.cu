#include "hip/hip_runtime.h"
#define GLM_ENABLE_EXPERIMENTAL
typedef unsigned int uint;

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <glm/gtc/type_ptr.hpp>  
#include <vector>
#include "gpu_types.h"
#include <iostream>
#include <chrono>
#include <hip/hip_math_constants.h>

#include "obj_loader.h"
#include "utils.h"
#include "Object.h"
#include "camera.h"
#include "scene.h"

#include <algorithm> 
#include <stdexcept>  
#include <cmath>      
#include <cfloat>    
#include <vector>     

__device__ __forceinline__ float3 mix(const float3& a, const float3& b, float t)
{
    return make_float3(a.x + (b.x - a.x) * t,
        a.y + (b.y - a.y) * t,
        a.z + (b.z - a.z) * t);
}

__device__ uint wangHash(uint s) {
    s = (s ^ 61u) ^ (s >> 16);
    s *= 9u;  s = s ^ (s >> 4);  s *= 0x27d4eb2du;
    s = s ^ (s >> 15); return s;
}

__device__ float rand(uint& state)
{
    state = wangHash(state);  return state * 2.3283064365387e-10f;
}

__host__ __device__ inline float3  operator+(const float3& a,
    const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ inline float3  operator-(const float3& a,
    const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ inline float3  operator*(const float3& a, const float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ inline float3  operator*(const float b, const float3& a)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ inline float3  operator/(const float3& a, const float b)
{
    float inv = 1.0f / b;  return make_float3(a.x * inv, a.y * inv, a.z * inv);
}

__host__ __device__ inline float3  operator*(const float3& a,
    const float3& b)
{
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}


struct GPU_Ray { float3 orig, dir; };
__device__ inline float3  cross(const float3& a, const float3& b) { return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x); }
__device__ inline float   dot3(const float3& a, const float3& b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
__device__ inline float3  normalize3(float3 v) { float inv = rsqrtf(dot3(v, v)); return make_float3(v.x * inv, v.y * inv, v.z * inv); }

__device__ __forceinline__ float3 clamp01(float3 v)
{
    return make_float3(fminf(fmaxf(v.x, 0.f), 1.f),
        fminf(fmaxf(v.y, 0.f), 1.f),
        fminf(fmaxf(v.z, 0.f), 1.f));
}

__device__ float3 sampleHemisphereCosine(float3 N, float r1, float r2)
{
    float phi = 2.f * HIP_PI_F * r1;
    float cosT = sqrtf(1.f - r2);
    float sinT = sqrtf(r2);
    float3 T, B;
    if (fabsf(N.z) < .999f) T = normalize3(cross(make_float3(0, 0, 1), N));
    else                    T = normalize3(cross(make_float3(0, 1, 0), N));
    B = cross(N, T);
    return normalize3(cosT * N + cosf(phi) * sinT * T + sinf(phi) * sinT * B);
}

GPUTriangle* d_tris = nullptr;
int32_t* d_tri_indices = nullptr;
GPUBVHNode* d_bvh_nodes = nullptr;
GPUMaterial* d_materials = nullptr;
GPUObject* d_objects = nullptr;
GPU_AABB host_global_bb;

static uchar4* d_fb = nullptr;

GPULightSphere* d_lights = nullptr;

static float3* d_accum = nullptr;

static unsigned int frameID = 0;

std::vector<GPUTriangle> all_tris;
std::vector<int32_t>     all_tri_indices;
std::vector<GPUBVHNode>  all_bvh_nodes;
std::vector<GPUMaterial> all_materials;
std::vector<GPUObject>   all_objects;

GPUCamera* d_cam;

std::vector<GPULightSphere> h_lights;

__device__ __forceinline__ float  clamp01(float v)
{
    return fminf(fmaxf(v, 0.f), 1.f);
}

__device__ bool hitAABB(const GPU_AABB& b, const GPU_Ray& r, float tMin, float tMax)
{
    float3 invD = make_float3(1.0f / r.dir.x, 1.0f / r.dir.y, 1.0f / r.dir.z);
    float3 t0 = (b.min - r.orig) * invD;
    float3 t1 = (b.max - r.orig) * invD;
    float3 tsm = make_float3(fminf(t0.x, t1.x), fminf(t0.y, t1.y), fminf(t0.z, t1.z));
    float3 tbg = make_float3(fmaxf(t0.x, t1.x), fmaxf(t0.y, t1.y), fmaxf(t0.z, t1.z));
    tMin = fmaxf(tMin, fmaxf(tsm.x, fmaxf(tsm.y, tsm.z)));
    tMax = fminf(tMax, fminf(tbg.x, fminf(tbg.y, tbg.z)));
    return tMax >= tMin;
}

__device__ bool hitTriangle(const GPUTriangle& tri, const GPU_Ray& r, float& t, float3& N)
{
    const float3 e1 = tri.v1 - tri.v0;
    const float3 e2 = tri.v2 - tri.v0;
    const float3 p = cross(r.dir, e2);
    const float  det = dot3(e1, p);
    if (fabsf(det) < 1e-8f) return false;
    const float inv = 1.0f / det;
    const float3 s = r.orig - tri.v0;
    const float  u = dot3(s, p) * inv; if (u < 0 || u > 1) return false;
    const float3 q = cross(s, e1);
    const float  v = dot3(r.dir, q) * inv; if (v < 0 || u + v > 1) return false;
    const float  tt = dot3(e2, q) * inv;     if (tt < 0.001f)      return false;
    t = tt;  N = normalize3(tri.n);
    return true;
}

__global__ void render_kernel(float3* accum,const GPUCamera* cam, const GPULightSphere* lights, int lightCnt, uint frameID, uchar4* fb,
    int                width,
    int                height,

    const GPUTriangle* tris,
    const int32_t* triIndices,
    const GPUBVHNode* nodes,
    const GPUMaterial* mats,
    const GPUObject* objects,
    int                objectCount,

    const GPU_AABB         globalBB)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) return;


    uint seed = ((py * width + px) << 16) ^ frameID;
    GPU_Ray ray;
    ray.orig = cam->pos;

    float sx = ((px + 0.5f) / width * 2.f - 1.f);
    float sy = ((py + 0.5f) / height * 2.f - 1.f);

    ray.dir = normalize3(
        cam->dir * cam->focal_len +
        cam->right * sx +
        cam->up * sy);


    float  bestT = 1e30f;
    float3 bestN = make_float3(0, 0, 0);
    GPUMaterial bestMat; bestMat.albedo = make_float3(1.0f, 1.0f, 1.0f);

    for (int o = 0; o < objectCount; ++o)
    {
        const GPUObject& obj = objects[o];
        int  stack[32];   int stackPtr = 0;
        stack[stackPtr++] = obj.node_offset;
        while (stackPtr)
        {
            int nodeIdx = stack[--stackPtr];
            const GPUBVHNode& n = nodes[nodeIdx];
            if (!hitAABB(n.bbox, ray, 0.001f, bestT)) continue;

            if (n.left == -1) {
                for (int i = 0; i < n.count; ++i) {
                    int triID = obj.tri_offset + triIndices[n.start + i];
                    float  t; float3 N;
                    if (hitTriangle(tris[triID], ray, t, N) && t < bestT) {
                        if (dot3(N, ray.dir) > 0.0f) N = N * -1.0f;

                        bestT = t;
                        bestN = N;
                        bestMat = mats[obj.mat_offset];
                    }
                }
            }
            else {
                stack[stackPtr++] = n.left;
                stack[stackPtr++] = n.right;
            }
        }
    }

    uchar4 out;



    float3 throughput = make_float3(1.0f, 1.0f, 1.0f);   
    float3 radiance = make_float3(0.0f, 0.0f, 0.0f);   

    for (int depth = 0; depth < 8; ++depth)
    {
    
        float  bestT = 1e30f;
        float3 bestN = make_float3(0.0f, 0.0f, 0.0f);
        GPUMaterial bestMat; bestMat.albedo = make_float3(1.0f, 1.0f, 1.0f);

        {                               
            for (int o = 0; o < objectCount; ++o) {
                const GPUObject& obj = objects[o];
                int stack[32], sp = 0;           stack[sp++] = obj.node_offset;
                while (sp) {
                    const GPUBVHNode& N = nodes[stack[--sp]];
                    if (!hitAABB(N.bbox, ray, 1e-3f, bestT)) continue;
                    if (N.left == -1) {
                        for (int i = 0; i < N.count; ++i) {
                            int id = obj.tri_offset + triIndices[N.start + i];
                            float t; float3 Nn;
                            if (hitTriangle(tris[id], ray, t, Nn) && t < bestT) {
                                if (dot3(Nn, ray.dir) > 0) Nn = Nn * -1;
                                bestT = t; bestN = Nn; bestMat = mats[obj.mat_offset];
                            }
                        }
                    }
                    else { stack[sp++] = N.left; stack[sp++] = N.right; }
                }
            }
        }

        if (bestT > 1e29f) {
            float  t = clamp01(ray.dir.y * .5f + .5f);
            const float3 skyTop = make_float3(0.65f, 0.80f, 1.0f);
            const float3 skyBottom = make_float3(1.00f, 1.00f, 1.00f);
            radiance = radiance+ throughput * mix(skyBottom, skyTop, t);
            break;
        }

       
        float3 hitP = ray.orig + bestT * ray.dir + bestN * 1e-3f;
        float3 Ldir = normalize3(make_float3(-0.2f, 0.55f, -0.4f));   
        float  nDotL = fmaxf(0.f, dot3(bestN, Ldir));

        bool inShadow = false;    
        if (!inShadow && nDotL > 0) {
            const float3 Li = make_float3(25.f, 25.f, 25.f);                 
            radiance = radiance + throughput * Li * nDotL;
        }

        float3 newDir = sampleHemisphereCosine(bestN, rand(seed), rand(seed));
        float  cosTheta = fmaxf(0.f, dot3(bestN, newDir));
        throughput = throughput* bestMat.albedo * (cosTheta / HIP_PI_F);

        
        float q = fmaxf(throughput.x, fmaxf(throughput.y, throughput.z));
        if (rand(seed) > q) break;
        throughput = throughput / q;

        ray.orig = hitP;
        ray.dir = newDir;
    }

    uint idxPix = py * width + px;
    float3 old = accum[idxPix];
    uint   sampleCount = frameID + 1;       
    accum[idxPix] = (old * frameID + radiance) / sampleCount;
    float exposure = 1.5f; 
    float3 mapped = accum[idxPix] * exposure;     
    mapped = make_float3(mapped.x / (1.0f + mapped.x),
        mapped.y / (1.0f + mapped.y),
        mapped.z / (1.0f + mapped.z));
    mapped = clamp01(mapped);                    
    fb[idxPix] = make_uchar4(mapped.x * 255, mapped.y * 255, mapped.z * 255, 255);

}


GLuint tex;
hipGraphicsResource* cuda_tex_resource;
int width = 1280, height = 720;

// Setup GL texture
void createTexture() {
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0,
        GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);

    
    hipGraphicsGLRegisterImage(&cuda_tex_resource, tex,
        GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
}

void setupScene() {
   
    //main defn 
    Camera c1(8.0f, 35.0f, height, width);
    Scene s1(c1);
    material m;
    m.albedo = glm::vec3(1.0f, 0.0f, 0.0f);
    m.metallic = 0.0f;
    m.roughness = 1.0f;
    glm::vec3 rotEuler = glm::vec3(glm::radians(0.0f),
        glm::radians(0.0f),
        0.0f);
    Object o1("street_new.obj", glm::vec3(0.0f, -15.0f, -125.0f), rotEuler, m);
    o1.BuildBVH(0, o1.Tri_index.size());
    s1.addObj(o1);
    s1.gen_global_BB();

    //send data to GPU

    GPULightSphere sun;
    sun.centre = make_float3(-50, 470, -90);
    sun.radius = 0.55f;
    sun.colour = make_float3(1.0f, 0.96f, 0.86f);
    sun.intensity = 1.5e1;
    h_lights.push_back(sun);


   
    size_t tri_off = 0,
        idx_off = 0,
        node_off = 0,
        mat_off = 0;

    int matid = 0;
    for (auto& obj : s1.ObjectList) {

        for (auto& t : obj.Tri_data) {
            GPUTriangle gt;
            gt.v0 = { t.v1.x, t.v1.y, t.v1.z };
            gt.v1 = { t.v2.x, t.v2.y, t.v2.z };
            gt.v2 = { t.v3.x, t.v3.y, t.v3.z };
            gt.n = { t.n.x, t.n.y, t.n.z };
            gt.material_id = matid;
            all_tris.push_back(gt);
        }

        for (int i : obj.Tri_index)
            all_tri_indices.push_back(i);


        for (auto& n : obj.BVH_Tree) {
            GPUBVHNode gn;
            gn.bbox.min = { n.box.min.x, n.box.min.y, n.box.min.z };
            gn.bbox.max = { n.box.max.x, n.box.max.y, n.box.max.z };
            gn.left = n.left;
            gn.right = n.right;
            gn.start = n.start;
            gn.count = n.count;
            all_bvh_nodes.push_back(gn);
        }


        m = obj.m;
        GPUMaterial gm;
        gm.albedo = { m.albedo.r, m.albedo.g, m.albedo.b };
        gm.roughness = m.roughness;
        gm.metallic = m.metallic;
        all_materials.push_back(gm);

        GPUObject go;
        go.tri_offset = tri_off;
        go.tri_count = (int)obj.Tri_index.size();
        go.node_offset = node_off;
        go.node_count = (int)obj.BVH_Tree.size();
        go.mat_offset = matid;


        all_objects.push_back(go);


        tri_off += obj.Tri_data.size();
        idx_off += obj.Tri_index.size();
        node_off += obj.BVH_Tree.size();
        mat_off += 1;

        matid++;
    }

    
    host_global_bb.min = { s1.global_bb.min.x,
                           s1.global_bb.min.y,
                           s1.global_bb.min.z };
    host_global_bb.max = { s1.global_bb.max.x,
                           s1.global_bb.max.y,
                           s1.global_bb.max.z };


    hipMalloc(&d_tris, all_tris.size() * sizeof(GPUTriangle));
    hipMalloc(&d_tri_indices, all_tri_indices.size() * sizeof(int32_t));
    hipMalloc(&d_bvh_nodes, all_bvh_nodes.size() * sizeof(GPUBVHNode));
    hipMalloc(&d_materials, all_materials.size() * sizeof(GPUMaterial));
    hipMalloc(&d_objects, all_objects.size() * sizeof(GPUObject));
    hipMalloc(&d_fb, width * height * sizeof(uchar4));
    hipMalloc(&d_accum, width * height * sizeof(float3));


    hipMemcpy(d_tris, all_tris.data(), all_tris.size() * sizeof(GPUTriangle), hipMemcpyHostToDevice);
    hipMemcpy(d_tri_indices, all_tri_indices.data(), all_tri_indices.size() * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bvh_nodes, all_bvh_nodes.data(), all_bvh_nodes.size() * sizeof(GPUBVHNode), hipMemcpyHostToDevice);
    hipMemcpy(d_materials, all_materials.data(), all_materials.size() * sizeof(GPUMaterial), hipMemcpyHostToDevice);
    hipMemcpy(d_objects, all_objects.data(), all_objects.size() * sizeof(GPUObject), hipMemcpyHostToDevice);


    GPUCamera h_cam;
    h_cam.pos = make_float3(0, 1.0f, 3.0f);

    glm::vec3 fwd = glm::normalize(glm::vec3(0, 0, -1));
    glm::vec3 right = glm::normalize(glm::cross(fwd, { 0,1,0 }));
    glm::vec3 up = glm::cross(right, fwd);

    h_cam.dir = make_float3(fwd.x, fwd.y, fwd.z);
    h_cam.right = make_float3(right.x, right.y, right.z);
    h_cam.up = make_float3(up.x, up.y, up.z);
    h_cam.focal_len = 1.0f;
    h_cam.lens_radius = h_cam.focal_len * tanf(glm::radians(0.5f)); 

    hipMalloc(&d_cam, sizeof(GPUCamera));
    hipMemcpy(d_cam, &h_cam, sizeof(GPUCamera), hipMemcpyHostToDevice);

    
    hipMalloc(&d_lights, h_lights.size() * sizeof(GPULightSphere));
    hipMemcpy(d_lights, h_lights.data(),
        h_lights.size() * sizeof(GPULightSphere),
        hipMemcpyHostToDevice);

    hipMalloc(&d_accum, width* height * sizeof(float3));
    hipMemset(d_accum, 0, width* height * sizeof(float3));

}


float runCuda() {
    hipArray_t array;
    hipGraphicsMapResources(1, &cuda_tex_resource);
    hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0);

    hipResourceDesc desc{};
    desc.resType = hipResourceTypeArray;
    desc.res.array.array = array;
    
    //end def
    dim3 block(32, 16);
    dim3 grid((width + block.x - 1) / block.x,
        (height + block.y - 1) / block.y);

    // CUDA Timing
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    render_kernel << <grid, block >> > (
        d_accum,
        d_cam,
        d_lights,
        h_lights.size(),
        frameID++,
        d_fb,
        width, height,
        d_tris,
        d_tri_indices,
        d_bvh_nodes,
        d_materials,
        d_objects,
        static_cast<int>(all_objects.size()),
        host_global_bb);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy2DToArray(array, 0, 0,
        d_fb, width * sizeof(uchar4),
        width * sizeof(uchar4), height,
        hipMemcpyDeviceToDevice);

    hipGraphicsUnmapResources(1, &cuda_tex_resource);

    hipEventDestroy(start);
    hipEventDestroy(stop);

   

    return milliseconds;
}


void drawTexture() {
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, tex);
    glBegin(GL_QUADS);
 
    glTexCoord2f(0, 1);  glVertex2f(-1, 1);


    glTexCoord2f(1, 1);  glVertex2f(1, 1);


    glTexCoord2f(1, 0);  glVertex2f(1, -1);

    glTexCoord2f(0, 0);  glVertex2f(-1, -1);
    glEnd();
}

int main() {
    if (!glfwInit()) return -1;

    GLFWwindow* window = glfwCreateWindow(width, height, "My Raytracer", nullptr, nullptr);
    if (!window) return -1;
    glfwMakeContextCurrent(window);

    glewInit();
    createTexture();

    
    using Clock = std::chrono::high_resolution_clock;
    auto lastTime = Clock::now();
    int frameCount = 0;
    setupScene();
    while (!glfwWindowShouldClose(window)) {
        auto frameStart = Clock::now();

        float kernelTime = runCuda();

        glClear(GL_COLOR_BUFFER_BIT);
        drawTexture();

        glfwSwapBuffers(window);
        glfwPollEvents();

        frameCount++;
        auto now = Clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(now - lastTime).count();
        if (elapsed >= 1000) {
            std::cout << "FPS: " << frameCount << " | Kernel time: " << kernelTime << " ms" << std::endl;
            frameCount = 0;
            lastTime = now;
        }
    }

    hipGraphicsUnregisterResource(cuda_tex_resource);
    glDeleteTextures(1, &tex);
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
